#include "hip/hip_runtime.h"
#include <stdio.h>
#include <ATen/ATen.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

__global__ void NmDistanceKernel(int b,int n, const int dim,const float * xyz,int m,const float * xyz2,float * result,int * result_i){
	const int batch=512;
	__shared__ float buf[batch*16];
    assert( dim <= 16 );
	for (int i=blockIdx.x;i<b;i+=gridDim.x){
		for (int k2=0;k2<m;k2+=batch){
			int end_k=min(m,k2+batch)-k2;
			for (int j=threadIdx.x;j<end_k*dim;j+=blockDim.x){
				buf[j]=xyz2[(i*m+k2)*dim+j];
			}
			__syncthreads();
			for (int j=threadIdx.x+blockIdx.y*blockDim.x;j<n;j+=blockDim.x*gridDim.y){
				const float* xyz1= &(xyz[(i*n+j)*dim]) ;
				int best_i=0;
				float best=0;
				int end_ka=end_k-(end_k&3);
				if (end_ka==batch){
					for (int k=0;k<batch;k+=4){
						{
                            float d = 0.0;
                            for(int di=0;di<dim;++di)
							{
                                float dif=buf[k*dim+di]-xyz1[di];
                                d += dif*dif;
                            }
							if (k==0 || d<best){
								best=d;
								best_i=k+k2;
							}
						}
						{
                            float d = 0.0;
                            for(int di=0;di<dim;++di)
							{
                                float dif=buf[(k+1)*dim+di]-xyz1[di];
                                d += dif*dif;
                            }
							if (d<best){
								best=d;
								best_i=k+k2+1;
							}
						}
						{
                            float d = 0.0;
                            for(int di=0;di<dim;++di)
							{
                                float dif=buf[(k+2)*dim+di]-xyz1[di];
                                d += dif*dif;
                            }
							if (d<best){
								best=d;
								best_i=k+k2+2;
							}
						}
						{
                            float d = 0.0;
                            for(int di=0;di<dim;++di)
							{
                                float dif=buf[(k+3)*dim+di]-xyz1[di];
                                d += dif*dif;
                            }
							if (d<best){
								best=d;
								best_i=k+k2+3;
							}
						}
					}
				}else{
					for (int k=0;k<end_ka;k+=4){
						{
                            float d = 0.0;
                            for(int di=0;di<dim;++di)
							{
                                float dif=buf[k*dim+di]-xyz1[di];
                                d += dif*dif;
                            }
							if (k==0 || d<best){
								best=d;
								best_i=k+k2;
							}
						}
						{
                            float d = 0.0;
                            for(int di=0;di<dim;++di)
							{
                                float dif=buf[(k+1)*dim+di]-xyz1[di];
                                d += dif*dif;
                            }
							if (d<best){
								best=d;
								best_i=k+k2+1;
							}
						}
						{
                            float d = 0.0;
                            for(int di=0;di<dim;++di)
							{
                                float dif=buf[(k+2)*dim+di]-xyz1[di];
                                d += dif*dif;
                            }
							if (d<best){
								best=d;
								best_i=k+k2+2;
							}
						}
						{
                            float d = 0.0;
                            for(int di=0;di<dim;++di)
							{
                                float dif=buf[(k+3)*dim+di]-xyz1[di];
                                d += dif*dif;
                            }
							if (d<best){
								best=d;
								best_i=k+k2+3;
							}
						}
					}
				}
				for (int k=end_ka;k<end_k;k++){
                    float d = 0.0;
                    for(int di=0;di<dim;++di)
                    {
                        float dif=buf[k*dim+di]-xyz1[di];
                        d += dif*dif;
                    }
					if (k==0 || d<best){
						best=d;
						best_i=k+k2;
					}
				}
				if (k2==0 || result[(i*n+j)]>best){
					result[(i*n+j)]=best;
					result_i[(i*n+j)]=best_i;
				}
			}
			__syncthreads();
		}
	}
}
// int chamfer_cuda_forward(int b,int n,const float * xyz,int m,const float * xyz2,float * result,int * result_i,float * result2,int * result2_i, hipStream_t stream){
int chamfer_cuda_forward(at::Tensor xyz1, at::Tensor xyz2, at::Tensor dist1, at::Tensor dist2, at::Tensor idx1, at::Tensor idx2){

	const auto batch_size = xyz1.size(0);
	const auto n = xyz1.size(1); //num_points point cloud A
	const auto m = xyz2.size(1); //num_points point cloud B
    const auto dim = xyz1.size(2);
    if( dim != xyz2.size(2) ){
        printf("dim do not match in chamfer_cuda_forward\n");
        return 0;
    }

	NmDistanceKernel<<<dim3(32,16,1),512>>>(batch_size, n, dim, xyz1.data<float>(), m, xyz2.data<float>(), dist1.data<float>(), idx1.data<int>());
	NmDistanceKernel<<<dim3(32,16,1),512>>>(batch_size, m, dim, xyz2.data<float>(), n, xyz1.data<float>(), dist2.data<float>(), idx2.data<int>());

	hipError_t err = hipGetLastError();
	  if (err != hipSuccess) {
	    printf("error in nnd updateOutput: %s\n", hipGetErrorString(err));
	    //THError("aborting");
	    return 0;
	  }
	  return 1;


}
__global__ void NmDistanceGradKernel(int b,int n,const int dim,const float * xyz1,int m,const float * xyz2,const float * grad_dist1,const int * idx1,float * grad_xyz1,float * grad_xyz2){
	for (int i=blockIdx.x;i<b;i+=gridDim.x){
		for (int j=threadIdx.x+blockIdx.y*blockDim.x;j<n;j+=blockDim.x*gridDim.y){
            int j2=idx1[i*n+j];
            float g=grad_dist1[i*n+j]*2;
            for(int di=0;di<dim;++di)
			{
                float x1=xyz1[(i*n+j)*dim+di];
                float x2=xyz2[(i*m+j2)*dim+di];
                atomicAdd(&(grad_xyz1[(i*n+j)*dim+di]),g*(x1-x2));
                atomicAdd(&(grad_xyz2[(i*m+j2)*dim+di]),-(g*(x1-x2)));
            }
		}
	}
}
// int chamfer_cuda_backward(int b,int n,const float * xyz1,int m,const float * xyz2,const float * grad_dist1,const int * idx1,const float * grad_dist2,const int * idx2,float * grad_xyz1,float * grad_xyz2, hipStream_t stream){
int chamfer_cuda_backward(at::Tensor xyz1, at::Tensor xyz2, at::Tensor gradxyz1, at::Tensor gradxyz2, at::Tensor graddist1, at::Tensor graddist2, at::Tensor idx1, at::Tensor idx2){
	// hipMemset(grad_xyz1,0,b*n*3*4);
	// hipMemset(grad_xyz2,0,b*m*3*4);
	
	const auto batch_size = xyz1.size(0);
	const auto n = xyz1.size(1); //num_points point cloud A
	const auto m = xyz2.size(1); //num_points point cloud B
    const auto dim = xyz1.size(2);
    if( dim != xyz2.size(2) ){
        printf("dim do not match in chamfer_cuda_forward\n");
        return 0;
    }

	NmDistanceGradKernel<<<dim3(1,16,1),256>>>(batch_size,n,dim,xyz1.data<float>(),m,xyz2.data<float>(),graddist1.data<float>(),idx1.data<int>(),gradxyz1.data<float>(),gradxyz2.data<float>());
	NmDistanceGradKernel<<<dim3(1,16,1),256>>>(batch_size,m,dim,xyz2.data<float>(),n,xyz1.data<float>(),graddist2.data<float>(),idx2.data<int>(),gradxyz2.data<float>(),gradxyz1.data<float>());
	
	hipError_t err = hipGetLastError();
	  if (err != hipSuccess) {
	    printf("error in nnd get grad: %s\n", hipGetErrorString(err));
	    //THError("aborting");
	    return 0;
	  }
	  return 1;
	
}

__device__ inline void swapf(float & a, float & b)
{   
    float tmp = a;
    a = b;
    b = tmp;
}

__device__ inline void swap(int & a, int & b)
{
    int tmp = a;
    a = b ;
    b = tmp;
}

__global__ void KnnKernel(int b,const int n,const int dim,const float * xyz,const int k,float * result,int * result_i){
    const int size = 4096;
    __shared__ float dist[size];
    __shared__ int idx[size];
    assert( n <= size );
    for ( int bi = blockIdx.x ; bi < b ; bi += gridDim.x )
    {
        for ( int i = blockIdx.y ;  i < n  ; i += gridDim.y )
        {
            for ( int j = threadIdx.x ; j < n ; j += blockDim.x )
            {
                if( i == j ){
                    dist[j] = 0;
                    idx[j]  = j;
                    continue;
                }
                float d = 0.0;
                for ( int di = 0 ; di < dim ; ++di )
                {
                    float dif = xyz[(bi*n+i)*dim+di] - xyz[(bi*n+j)*dim+di];
                    d += dif*dif;
                }
                dist[j] = d;
                idx[j] = j;
            }
            __syncthreads();
            //odd-even sort
	    int pownum = int(log2(float(n)));
	    if ( n != int(pow(double(2), double(pownum))) ){
            for ( int cnt = 0 ; cnt < ( n + 1 ) / 2 ; ++cnt )
            {
                for ( int j = 2*threadIdx.x + 1 ; j < n ; j += 2*blockDim.x )
                {
                    if ( dist[j] < dist[ j - 1 ] )
                    {
                        swapf(dist[j], dist[j-1]);
                        swap(idx[j], idx[j-1]);
                    }
                }
                __syncthreads();
                for ( int j = 2*threadIdx.x + 2 ; j < n ; j += 2*blockDim.x )
                {
                    if ( dist[j] < dist[ j - 1 ] )
                    {
                        swapf(dist[j], dist[j-1]);
                        swap(idx[j], idx[j-1]);
                    }
                }
                __syncthreads();
            }
	    }else{	
            //Bitonic Sort
            for (unsigned int t = 2; t <= n ; t *= 2)
            {
                // Bitonic merge:
                for (unsigned int j = t / 2; j>0; j /= 2)
                {	
			for (unsigned int tid = threadIdx.x ; tid < n ; tid += blockDim.x )
                    	{
				unsigned int ixj = tid ^ j;
                    		if (ixj > tid)
                    		{
                        		if ((tid & t) == 0)
                        		{
                            			if (dist[tid] > dist[ixj])
                            			{
                                			swapf(dist[tid], dist[ixj]);
                                			swap(idx[tid], idx[ixj]);
                            			}
                        		}
                        		else
                        		{
                            			if (dist[tid] < dist[ixj])
                            			{
                                			swapf(dist[tid], dist[ixj]);
                                			swap(idx[tid], idx[ixj]);
                            			}
                        		}
                    		}
                    		
			}
			__syncthreads();	
                }
            }
	    }
            __syncthreads();
            //copy result
            for ( int j = threadIdx.x ; j < k  ; j += blockDim.x )
            {
                result[(bi*n+i)*k+j] = dist[j+1];
                result_i[ ((bi*n+i)*k+j)*2+0 ] = bi;
                result_i[ ((bi*n+i)*k+j)*2+1 ] = idx[j+1];
            }
            
        }
    }
}

int knn_cuda(at::Tensor xyz,at::Tensor k,at::Tensor dist,at::Tensor idx)
{
    const auto bs = xyz.size(0);
	const auto n = xyz.size(1); //num_points point cloud
    const auto d = xyz.size(2);
    int k_ = k.data<int>()[0];
	KnnKernel<<<dim3(bs,16,1),512>>>(bs,n,d,xyz.data<float>(),k_,dist.data<float>(),idx.data<int>());
	hipError_t err = hipGetLastError();
	  if (err != hipSuccess) {
	    printf("error in nnd Knn: %s\n", hipGetErrorString(err));
	    return 0;
	  }
	  return 1;
}

__global__ void interpKernel(const int b, const int p,const int L,const int H,const int W,const float* z,const float* prob,int* idx,float* w,float* pout)
{
    float stepy = 1.0 / float(H - 1);
    float stepx = 1.0 / float(W - 1);
    for ( int bi = blockIdx.x ; bi < b ; bi += gridDim.x )
        for ( int pi = blockIdx.y; pi < p ; pi += gridDim.y )
            for ( int li = threadIdx.x; li < L ; li += blockDim.x )
            {
                float zx = z[((bi*p+pi)*2+0)*L+li];
                float zy = z[((bi*p+pi)*2+1)*L+li];
                if( zx < 0.0 || zy < 0.0 || zx >= 1.0 || zy >= 1.0 )
                {
                    pout[(bi*p+pi)*L+li] = 0.0;
                    for( int i = 0 ; i < 4 ; i ++)
                    {
                        idx[(((bi*p+pi)*2+0)*4+i)*L+li] = -1;
                        idx[(((bi*p+pi)*2+1)*4+i)*L+li] = -1;
                        w[((bi*p+pi)*4+i)*L+li] = 0.0;
                    }
                    continue;
                }
                int zxn = int(zx / stepx);
                int zyn = int(zy / stepy);
                //
                idx[(((bi*p+pi)*2+0)*4+0)*L+li] = zxn;
                idx[(((bi*p+pi)*2+1)*4+0)*L+li] = zyn;
                idx[(((bi*p+pi)*2+0)*4+1)*L+li] = zxn;
                idx[(((bi*p+pi)*2+1)*4+1)*L+li] = zyn+1;
                idx[(((bi*p+pi)*2+0)*4+2)*L+li] = zxn+1;
                idx[(((bi*p+pi)*2+1)*4+2)*L+li] = zyn;
                idx[(((bi*p+pi)*2+0)*4+3)*L+li] = zxn+1;
                idx[(((bi*p+pi)*2+1)*4+3)*L+li] = zyn+1;
                //
                float x1w = zx - zxn*stepx;
                float x2w = (zxn+1)*stepx - zx;
                float y1w = zy - zyn*stepy;
                float y2w = (zyn+1)*stepy - zy;
                //
                float w1 = y2w*x2w/((y1w+y2w)*(x1w+x2w));
                w[((bi*p+pi)*4+0)*L+li] = w1;
                float w2 = y1w*x2w/((y1w+y2w)*(x1w+x2w));
                w[((bi*p+pi)*4+1)*L+li] = w2;
                float w3 = y2w*x1w/((y1w+y2w)*(x1w+x2w));
                w[((bi*p+pi)*4+2)*L+li] = w3;
                float w4 =  y1w*x1w/((y1w+y2w)*(x1w+x2w));
                w[((bi*p+pi)*4+3)*L+li] = w4;
                //
                float p1 = prob[((bi*p+pi)*H+zyn)*W+zxn];
                float p2 = prob[((bi*p+pi)*H+zyn+1)*W+zxn];
                float p3 = prob[((bi*p+pi)*H+zyn)*W+zxn+1];
                float p4 = prob[((bi*p+pi)*H+zyn+1)*W+zxn+1];
                //
                pout[(bi*p+pi)*L+li] = p1*w1+p2*w2+p3*w3+p4*w4;
            }
}

int interp_cuda_forward(at::Tensor z,at::Tensor prob,at::Tensor idx,at::Tensor w,at::Tensor pout)
{
    const auto b = z.size(0);
    const auto p = z.size(1); 
    const auto L = z.size(3);
    const auto H = prob.size(-2);
    const auto W = prob.size(-1);
    interpKernel<<<dim3(b,25,1),512>>>(b,p,L,H,W,z.data<float>(),prob.data<float>(),idx.data<int>(),w.data<float>(),pout.data<float>());
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("error in interp_cuda_forward: %s\n", hipGetErrorString(err));
        return 0;
    }
    return 1;
}

__global__ void interpGradKernel(const int b, const int p,const int L,const int H,const int W,const float* grad,const int* idx,const float* w,float* gradp)
{
    for ( int bi = blockIdx.x ; bi < b ; bi += gridDim.x )
        for ( int pi = blockIdx.y; pi < p ; pi += gridDim.y )
            for ( int li = threadIdx.x; li < L ; li += blockDim.x )
            {
                float g = grad[(bi*p+pi)*L+li];
                for(int i = 0 ; i < 4; i++)
                {
                    float wv = w[((bi*p+pi)*4+i)*L+li];
                    const int x = idx[(((bi*p+pi)*2+0)*4+i)*L+li];
                    const int y = idx[(((bi*p+pi)*2+1)*4+i)*L+li];
                    if((x == -1) || (y == -1))break;
                    atomicAdd(&(gradp[((bi*p+pi)*H+y)*W+x]),g*wv);
                }
            }
}

int interp_cuda_backward(at::Tensor grad,at::Tensor idx,at::Tensor w,at::Tensor gradp)
{
    const auto b = grad.size(0);
    const auto p = grad.size(1); 
    const auto L = grad.size(2);
    const auto H = gradp.size(-2);
    const auto W = gradp.size(-1);
    interpGradKernel<<<dim3(b,25,1),512>>>(b,p,L,H,W,grad.data<float>(),idx.data<int>(),w.data<float>(),gradp.data<float>());
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("error in interp_cuda_backward: %s\n", hipGetErrorString(err));
        return 0;
    }
    return 1;
}

__global__ void selectKernel(const int b, const int p,const int dim,const int L,const int N,const float* inz,const float* inp,const unsigned char* select,int* idx,float* outz,float* outp)
{
    for ( int bi = blockIdx.x ; bi < b ; bi += gridDim.x )
        for ( int pi = blockIdx.y; pi < p ; pi += gridDim.y )
        {
            for ( int ni = threadIdx.x; ni < N ; ni += blockDim.x )
            {
                idx[(bi*p+pi)*N+ni] = -1;
            }
            __syncthreads();
            for ( int li = threadIdx.x; li < L ; li += blockDim.x )
            {
                if( select[(bi*p+pi)*L+li] )
                {
                    for( int ni = 0 ; ni < N ; ni ++ )
                    {
                        int v = atomicExch(&(idx[(bi*p+pi)*N+ni]),li);
                        if( v == -1 )
                        {
                            for(int di=0;di<dim;++di)
                            {
                                outz[((bi*p+pi)*dim+di)*N+ni] = inz[((bi*p+pi)*dim+di)*L+li];
                            }
                            outp[(bi*p+pi)*N+ni] = inp[(bi*p+pi)*L+li];
                            break;
                        }else{
                            atomicExch(&(idx[(bi*p+pi)*N+ni]),v);
                        }
                    }
                }
            }
        }
}

int select_cuda_forward(at::Tensor z,at::Tensor pzv,at::Tensor select,at::Tensor idx,at::Tensor outz,at::Tensor outp)
{
    const auto b = z.size(0);
    const auto p = z.size(1); 
    const auto L = z.size(-1);
    const auto dim = z.size(2);
    const auto N = outz.size(-1);
    selectKernel<<<dim3(b,25,1),512>>>(b,p,dim,L,N,z.data<float>(),pzv.data<float>(),select.data<unsigned char>(),idx.data<int>(),outz.data<float>(),outp.data<float>());
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("error in select_cuda_forward: %s\n", hipGetErrorString(err));
        return 0;
    }
    return 1;
}

__global__ void selectGradKernel(const int b, const int p,const int dim,const int L,const int N,const float* outzgrad,const float* outpgrad,const int* idx,float* inzgrad,float* inpgrad)
{
    for ( int bi = blockIdx.x ; bi < b ; bi += gridDim.x )
        for ( int pi = blockIdx.y; pi < p ; pi += gridDim.y )
        {
            for ( int ni = threadIdx.x; ni < N ; ni += blockDim.x )
            {
                int li = idx[(bi*p+pi)*N+ni];
                for(int di=0;di<dim;++di)
                {
                    inzgrad[((bi*p+pi)*dim+di)*L+li] = outzgrad[((bi*p+pi)*dim+di)*N+ni];
                }
                inpgrad[(bi*p+pi)*L+li] = outpgrad[(bi*p+pi)*N+ni];
            }
        }
}

int select_cuda_backward(at::Tensor outzgrad,at::Tensor outpgrad,at::Tensor idx,at::Tensor inzgrad,at::Tensor inpgrad)
{
    const auto b = outzgrad.size(0);
    const auto p = outzgrad.size(1); 
    const auto N = outzgrad.size(-1);
    const auto d = outzgrad.dim();
    const auto L = inzgrad.size(-1);
    const auto dim = outzgrad.size(2);
    selectGradKernel<<<dim3(b,25,1),512>>>(b,p,dim,L,N,outzgrad.data<float>(),outpgrad.data<float>(),idx.data<int>(),inzgrad.data<float>(),inpgrad.data<float>());
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("error in select_cuda_backward: %s\n", hipGetErrorString(err));
        return 0;
    }
    return 1;
}

